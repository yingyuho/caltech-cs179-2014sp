#include "hip/hip_runtime.h"
// -*- C++ -*-
#include <cstdio>

#include <hip/hip_runtime.h>

#include "Main1_cuda.cuh"

//since we can't really dynamically size this array,
//let's leave its size at the default polynomial order
__constant__ float constant_c[10];


__global__
void
cudaSum_atomic_kernel(const float* const inputs,
                                     unsigned int numberOfInputs,
                                     const float* const c,
                                     unsigned int polynomialOrder,
                                     float* output) {

    float partialSum = 0.0;

    unsigned int inputIndex = blockIdx.x * blockDim.x + threadIdx.x;

    unsigned int i;
    float p, r;

    if (polynomialOrder == 0)
       return;

    while (inputIndex < numberOfInputs) {
      partialSum += c[0];

      p = 1.0;
      r = inputs[inputIndex];
      for (i = 1; i < polynomialOrder; ++i) {
        partialSum += c[i] * (p *= r);
      }

      inputIndex += blockDim.x * gridDim.x;
    }

    atomicAdd(output, partialSum);
}

__global__
void
cudaSum_linear_kernel(const float* const inputs, 
                                  unsigned int numberOfInputs, 
                                  const float* const c,
                                  unsigned int polynomialOrder, 
                                  float * output) {
    extern __shared__ float partial_outputs[];

    // Initialize shared memory to 0.0
    // if (threadIdx.x == 0) {
    //   for (unsigned int i = 0; i < blockDim.x; ++i)
    //     partial_outputs[i] = 0.0;
    // }

    // syncthreads();

    unsigned int inputIndex = blockIdx.x * blockDim.x + threadIdx.x;
    float * const pout = partial_outputs + threadIdx.x;

    *pout = 0;

    if (polynomialOrder == 0)
       return;

    while (inputIndex < numberOfInputs) {
      *pout += c[0];

      float p = 1.0;
      float r = inputs[inputIndex];
      
      for (unsigned int i = 1; i < polynomialOrder; ++i) {
        *pout += c[i] * (p *= r);
      }

      inputIndex += blockDim.x * gridDim.x;
    }

    syncthreads();

    // Accumulate results from shared memory
    if (threadIdx.x == 0) {
      float partialSum = 0.0;
      for (unsigned int i = 0; i < blockDim.x; ++i)
        partialSum += partial_outputs[i];
      atomicAdd(output, partialSum);
    }
}


/* Used in Assignment 2. Coming soon! */
__global__
void
cudaSum_divtree_kernel(const float* const inputs, 
                                  unsigned int numberOfInputs, 
                                  const float* const c,
                                  unsigned int polynomialOrder, 
                                  float * output) {
    

}

/* Used in Assignment 2. Coming soon! */
__global__
void
cudaSum_nondivtree_kernel(const float* const inputs, 
                                  unsigned int numberOfInputs, 
                                  const float* const c,
                                  unsigned int polynomialOrder, 
                                  float * output) {
    

}

/* Used in Assignment 2. Coming soon! */
__global__
void
cudaSum_constmem_kernel(const float* const inputs, 
                                  unsigned int numberOfInputs,
                                  unsigned int polynomialOrder, 
                                  float * output) {
    

}




void
cudaSumPolynomials(const float* const input,
                            const size_t numberOfInputs,
                            const float* const c,
                            const size_t polynomialOrder,
                            const Style style,
                            const unsigned int maxBlocks,
                            float * const output) {


    //Input values (your "r" values) go here on the GPU
    float *dev_input;
    
    //Your polynomial coefficients go here (GPU)
    float *dev_c;
    
    //Your output will go here (GPU)
    float *dev_output;
    const float float_zero = 0.0f;

    // Allocate memory for GPU to hold inputs
    hipMalloc((void **) &dev_input,  numberOfInputs  * sizeof(float));
    hipMemcpy(dev_input, input,  numberOfInputs  * sizeof(float), hipMemcpyHostToDevice);

    hipMalloc((void **) &dev_c,      polynomialOrder * sizeof(float));
    hipMemcpy(dev_c,     c,      polynomialOrder * sizeof(float), hipMemcpyHostToDevice);

    // Allocate memory for GPU to hold output
    hipMalloc((void **) &dev_output, sizeof(float));
    hipMemcpy(dev_output, &float_zero, sizeof(float), hipMemcpyHostToDevice);
    
    const unsigned int threadsPerBlock = 512;
    const unsigned int blocks 
                = min((float)maxBlocks, 
                        ceil(numberOfInputs/(float)threadsPerBlock));
    

    if (style == mutex) {
        cudaSum_atomic_kernel<<<blocks, threadsPerBlock>>>(
                dev_input, numberOfInputs, dev_c, polynomialOrder, dev_output);
    } else if (style == linear) {
        cudaSum_linear_kernel<<<blocks, threadsPerBlock, 
                threadsPerBlock*sizeof(float)>>>(dev_input, numberOfInputs, 
                dev_c, polynomialOrder, dev_output);
    } else if (style == divtree) {
        cudaSum_divtree_kernel<<<blocks, threadsPerBlock, 
                threadsPerBlock*sizeof(float)>>>(dev_input, numberOfInputs, 
                dev_c, polynomialOrder, dev_output);
    } else if (style == nondivtree) {
        cudaSum_nondivtree_kernel<<<blocks, threadsPerBlock, 
                threadsPerBlock*sizeof(float)>>>(dev_input, numberOfInputs, 
                dev_c, polynomialOrder, dev_output);
    } else if (style == constmem) {
        
        //initialize the constant memory
        hipMemcpyToSymbol(HIP_SYMBOL("constant_c"), c, polynomialOrder * sizeof(float),
                0, hipMemcpyHostToDevice);
        
        cudaSum_constmem_kernel<<<blocks, threadsPerBlock, 
                threadsPerBlock*sizeof(float)>>>(dev_input, numberOfInputs, 
                polynomialOrder, dev_output);
    } else {
        printf("Unknown style\n");
    }

    // Copy output from GPU
    hipMemcpy(output, dev_output, sizeof(float), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(dev_input);
    hipFree(dev_c);
    hipFree(dev_output);
}
